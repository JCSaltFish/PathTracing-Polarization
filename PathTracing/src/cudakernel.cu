#include "hip/hip_runtime.h"
#include <glm/gtx/constants.hpp>

#include <hiprand/hiprand_kernel.h>

#include "cudakernel.cuh"

__constant__ hiprandState_t* state;

__constant__ float PI;
__constant__ float EPSILON = 0.001f;

__constant__ int resX, resY;
__constant__ int maxDepth;
__constant__ int samples;

__constant__ float _camPos[3], _camDir[3], _camUp[3];
__constant__ float camFocal, camFovy;

__device__ GPUBVHNode* bvh = 0;
__constant__ int bvhSize = 0;

struct GPUImage
{
	int width;
	int height;
	float* data;
	__host__ __device__ GPUImage() :
		width(0),
		height(0),
		data(0)
	{
	}
};

__device__ GPUImage* textures = 0;
int numTextures = 0;

struct GPUPolarInfo
{
	glm::vec3 normal;
	glm::vec3 indir;
	glm::vec3 outdir;

	glm::vec2 R;
	glm::vec3 S;

	__device__ GPUPolarInfo()
	{
		normal = glm::vec3();
		indir = glm::vec3();
		outdir = glm::vec3();

		R = glm::vec2();
		S = glm::vec3();
	}
};

__device__ glm::vec2 GetRsRp(float n1, float n2, float cos_i)
{
	cos_i = fabs(cos_i);
	float cos_t = sqrtf(1.0f - (n1 * n1) / (n2 * n2) * (1.0f - cos_i * cos_i));
	float rs = (n1 * cos_i - n2 * cos_t) / (n1 * cos_i + n2 * cos_t);
	float rp = (n2 * cos_i - n1 * cos_t) / (n1 * cos_t + n2 * cos_i);
	return glm::vec2(rs, rp);
}

__device__ glm::mat3 GetRotationMatrix(float phi)
{
	glm::mat3 M;
	float sin_2phi = sinf(2.0f * phi);
	float cos_2phi = cosf(2.0f * phi);
	M[0] = glm::vec3(1.0f, 0.0f, 0.0f);
	M[1] = glm::vec3(0.0f, cos_2phi, -sin_2phi);
	M[2] = glm::vec3(0.0f, sin_2phi, cos_2phi);
	return M;
}

__device__ glm::vec3 CalculatePolarResult
(
	const glm::vec3& initDir,
	GPUPolarInfo* polarInfoList,
	int listSize,
	hiprandState_t& state
)
{
	int i = listSize;
	if (i == 0)
		return glm::vec3(0.0f);

	i--;
	glm::vec3 res = glm::vec3(0.0f);
	glm::vec3 s = glm::normalize(glm::vec3
	(
		hiprand_uniform(&state) * 2.0f - 1.0f,
		hiprand_uniform(&state) * 2.0f - 1.0f,
		hiprand_uniform(&state) * 2.0f - 1.0f
	));
	float phi = 2.0f * PI * hiprand_uniform(&state);
	for (; i >= 0; i--)
	{
		glm::vec3 d = polarInfoList[i].outdir;
		glm::vec3 s_old = s;
		s = glm::normalize(glm::cross(polarInfoList[i].normal, d));
		//phi = acos(glm::dot(s, s_old));
		phi = acos(glm::dot(s, s_old)) * glm::sign(glm::dot(glm::cross(s_old, s), d));

		res = GetRotationMatrix(phi) * res;
		const float Rs = polarInfoList[i].R.x * polarInfoList[i].R.x;
		const float Rp = polarInfoList[i].R.y * polarInfoList[i].R.y;
		glm::mat3 Refl = glm::mat3
		(
			glm::vec3((Rs + Rp) / 2.0f, (Rs - Rp) / 2.0f, 0.0f),
			glm::vec3((Rs - Rp) / 2.0f, (Rs + Rp) / 2.0f, 0.0f),
			glm::vec3(0.0f, 0.0f, polarInfoList[i].R.x * polarInfoList[i].R.y)
		);
		glm::mat3 Trans = glm::mat3
		(
			glm::vec3(1.0f - (Rs + Rp) / 2.0f, (Rp - Rs) / 2.0f, 0.0f),
			glm::vec3((Rp - Rs) / 2.0f, 1.0f - (Rs + Rp) / 2.0f, 0.0f),
			glm::vec3(0.0f, 0.0f, sqrt((1.0f - Rs) * (1.0f - Rp)))
		);
		res = Refl * res + Trans * polarInfoList[i].S;
	}
	glm::vec3 s_cam = glm::normalize(glm::cross(glm::vec3(0.0f, 0.0f, 1.0f), initDir));
	//phi = M_PI * 0.5f; // acos(0)
	phi = acos(glm::dot(s_cam, s)) * glm::sign(glm::dot(glm::cross(s, s_cam), initDir));

	return GetRotationMatrix(phi) * res;
}

__global__ void InitCuRand(int seed)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= resX || y >= resY)
		return;

	hiprand_init(seed, x + y * resX, 0, &state[x + y * resX]);
}

void InitCUDA()
{
	float h_pi = glm::pi<float>();
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(PI), &h_pi, sizeof(float)));
	gpuErrchk(hipDeviceSetLimit(hipLimitStackSize, 1024 * 8));
}

void CUDASetResolution(int x, int y)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(resX), &x, sizeof(unsigned)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(resY), &y, sizeof(unsigned)));

	hiprandState_t* d_randState;
	gpuErrchk(hipMalloc((void**)&d_randState, x * y * sizeof(hiprandState_t)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(state), &d_randState, sizeof(d_randState)));

	srand(time(0));
	int seed = rand();
	dim3 blockDim(16, 16, 1), gridDim(x / blockDim.x + 1, y / blockDim.y + 1, 1);
	InitCuRand << < gridDim, blockDim >> > (seed);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void CUDASetTraceDepth(int depth)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(maxDepth), &depth, sizeof(unsigned)));
}

void CUDASetCamera(float* pos, float* dir, float* up)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_camPos), pos, sizeof(float) * 3));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_camDir), dir, sizeof(float) * 3));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_camUp), up, sizeof(float) * 3));
}

void CUDASetProjection(float f, float fovy)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(camFocal), &f, sizeof(float)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(camFovy), &fovy, sizeof(float)));
}

void CUDASetBVH(GPUBVHNode* nodes, int size)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bvhSize), &size, sizeof(unsigned)));

	BVHNode* d_Nodes;
	gpuErrchk(hipMalloc((void**)&d_Nodes, size * sizeof(GPUBVHNode)));
	gpuErrchk(hipMemcpy(d_Nodes, nodes, size * sizeof(GPUBVHNode), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bvh), &d_Nodes, sizeof(GPUBVHNode*)));
}

void CUDALoadTextures(const std::vector<Image*>& texVec)
{
	int size = texVec.size();
	numTextures = size;

	GPUImage* h_Imgs = new GPUImage[size];
	for (int i = 0; i < size; i++)
	{
		int w = texVec[i]->width();
		int h = texVec[i]->height();
		h_Imgs[i].width = w;
		h_Imgs[i].height = h;

		int size = w * h * 4 * sizeof(float);
		float* h_data = new float[size];
		memcpy(h_data, texVec[i]->data(), size);
		float* d_data;
		gpuErrchk(hipMalloc(&d_data, size));
		gpuErrchk(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));
		delete[] h_data;
		h_Imgs->data = d_data;
	}

	GPUImage* d_Imgs;
	gpuErrchk(hipMalloc(&d_Imgs, size * sizeof(GPUImage)));
	gpuErrchk(hipMemcpy(d_Imgs, h_Imgs, size * sizeof(GPUImage), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(textures), &d_Imgs, size * sizeof(GPUImage*)));
	delete[] h_Imgs;
}

__device__ glm::vec4 CUDATex2D(const GPUImage& image, const glm::vec2& uv)
{
	if (uv.x > 1.0f || uv.x < 0.0f || uv.y > 1.0f || uv.y < 0.0f)
		return glm::vec4(0.0f);

	int w = image.width;
	int h = image.height;

	glm::ivec2 coord = glm::ivec2(w * uv.x, h * uv.y);
	float* p = image.data + (4 * (coord.y * w + coord.x));

	return glm::vec4(p[0], p[1], p[2], p[3]);
}

__device__ bool IsSameSide(const glm::vec3& p1, const glm::vec3& p2, const glm::vec3& a, const glm::vec3& b)
{
	glm::vec3 ba = b - a;
	glm::vec3 cp1 = glm::cross(ba, (p1 - a));
	glm::vec3 cp2 = glm::cross(ba, (p2 - a));

	return (glm::dot(cp1, cp2) >= 0);
}

__device__ bool IsInside(const glm::vec3& p, const glm::vec3& a, const glm::vec3& b, const glm::vec3& c)
{
	return (IsSameSide(p, a, b, c) && IsSameSide(p, b, a, c) && IsSameSide(p, c, a, b));
}

__device__ bool IntersectBox(const glm::vec3& ro, const glm::vec3& rd, const glm::vec3& bMin, const glm::vec3& bMax)
{
	glm::vec3 tMin = (bMin - ro) / rd;
	glm::vec3 tMax = (bMax - ro) / rd;
	glm::vec3 t1 = glm::min(tMin, tMax);
	glm::vec3 t2 = glm::max(tMin, tMax);
	float tNear = glm::max(glm::max(t1.x, t1.y), t1.z);
	float tFar = glm::min(glm::min(t2.x, t2.y), t2.z);
	if (tNear >= tFar)
		return false;
	return true;
}

__device__ bool Hit(const glm::vec3& ro, const glm::vec3& rd, Triangle& triangleOut, float& distOut)
{
	if (bvhSize <= 1)
		return false;

	bool res = false;

	distOut = float(0xFFFF);

	GPUBVHNode* stack[64];
	GPUBVHNode** pStack = stack;
	*pStack++ = NULL;

	GPUBVHNode* currNode = bvh;
	int stackIndex = 1;
	do
	{
		if (IntersectBox(ro, rd, currNode->box.min, currNode->box.max))
		{
			if (currNode->rightOffset == -1) // leaf
			{
				if (glm::dot(rd, currNode->triangle.normal) != 0.0f)
				{
					float d = glm::dot((currNode->triangle.v1 - ro), currNode->triangle.normal) / glm::dot(rd, currNode->triangle.normal);
					if (d >= 0)
					{
						glm::vec3 p = ro + rd * d;
						if (IsInside(p, currNode->triangle.v1, currNode->triangle.v2, currNode->triangle.v3))
						{
							if (d < distOut)
							{
								distOut = d;
								triangleOut = currNode->triangle;
							}
							res = true;
						}
					}
				}
				currNode = *--pStack;
				stackIndex--;
			}
			else // interier
			{
				GPUBVHNode* left = &(bvh[currNode->nodeIndex + 1]);
				GPUBVHNode* right = &(bvh[currNode->nodeIndex + currNode->rightOffset]);
				currNode = left;
				*pStack++ = right;
				stackIndex++;
			}
		}
		else
		{
			currNode = *--pStack;
			stackIndex--;
		}
	} while (stackIndex > 0 && stackIndex < 64);

	return res;
}

__device__ glm::vec2 GetUV(const glm::vec3& p, const Triangle& t)
{
	glm::vec3 v2 = p - t.v1;
	float d20 = glm::dot(v2, t.barycentricInfo.v0);
	float d21 = glm::dot(v2, t.barycentricInfo.v1);

	float alpha = (t.barycentricInfo.d11 * d20 - t.barycentricInfo.d01 * d21) *
		t.barycentricInfo.invDenom;
	float beta = (t.barycentricInfo.d00 * d21 - t.barycentricInfo.d01 * d20) *
		t.barycentricInfo.invDenom;

	return (1.0f - alpha - beta) * t.uv1 + alpha * t.uv2 + beta * t.uv3;
}

__device__ glm::vec3 GetSmoothNormal(const glm::vec3& p, const Triangle& t)
{
	glm::vec3 v2 = p - t.v1;
	float d20 = glm::dot(v2, t.barycentricInfo.v0);
	float d21 = glm::dot(v2, t.barycentricInfo.v1);

	float alpha = (t.barycentricInfo.d11 * d20 - t.barycentricInfo.d01 * d21) *
		t.barycentricInfo.invDenom;
	float beta = (t.barycentricInfo.d00 * d21 - t.barycentricInfo.d01 * d20) *
		t.barycentricInfo.invDenom;

	glm::vec3 n = (1.0f - alpha - beta) * t.n1 + alpha * t.n2 + beta * t.n3;
	glm::vec3 res = glm::normalize(glm::vec3(n.x, -n.y, n.z));
	return glm::normalize(n);
}

__device__ glm::vec3 reflect(glm::vec3 I, glm::vec3 N)
{
	return I - N * glm::dot(N, I) * glm::vec3(2);
}

__device__ glm::vec3 Trace(glm::vec3 ro, glm::vec3 rd, int& depth, bool& inside, hiprandState_t& state,
	GPUPolarInfo* polarInfoList)
{
	float d = 0.0f;
	Triangle t;
	if (Hit(ro, rd, t, d))
	{
		Material& mat = t.material;
		glm::vec3 p = ro + rd * d;
		glm::vec2 uv = GetUV(p, t);
		glm::vec3 n = t.normal;
		if (t.smoothing)
			n = GetSmoothNormal(p, t);
		if (glm::dot(n, rd) > 0.0f)
			n = -n;
		if (mat.normalTexId != -1)
		{
			glm::mat3 TBN = glm::mat3(t.tangent, t.bitangent, n);
			glm::vec3 nt = glm::vec3(CUDATex2D(textures[mat.normalTexId], uv)) * 2.0f - 1.0f;
			if (nt.z < 0.0f)
				nt = glm::vec3(nt.x, nt.y, 0.0f);
			nt = glm::normalize(nt);
			n = glm::normalize(TBN * nt);
		}
		p += n * EPSILON;

		if (depth < maxDepth * 2)
		{
			depth++;
			// Russian Roulette Path Termination
			float prob = glm::min(0.95f, glm::max(glm::max(mat.baseColor.x, mat.baseColor.y), mat.baseColor.z));
			if (depth >= maxDepth)
			{
				if (fabs(hiprand_uniform(&state)) > prob)
					return mat.emissive * mat.emissiveIntensity;
			}

			glm::vec3 r = reflect(rd, n);
			glm::vec3 reflectDir = r;
			if (mat.type == MaterialType::SPECULAR)
				reflectDir = r;
			else if (mat.type == MaterialType::DIFFUSE)
			{
				// Monte Carlo Integration
				glm::vec3 u = glm::abs(n.x) < 1.0f - EPSILON ? glm::cross(glm::vec3(1.0f, 0.0f, 0.0f), n) : glm::cross(glm::vec3(1.0f), n);
				u = glm::normalize(u);
				glm::vec3 v = glm::normalize(glm::cross(u, n));
				float w = hiprand_uniform(&state), theta = hiprand_uniform(&state);
				// uniformly sampling on hemisphere
				reflectDir = w * cosf(2.0f * PI * theta) * u + w * sinf(2.0f * PI * theta) * v + glm::sqrt(1.0f - w * w) * n;
				reflectDir = glm::normalize(reflectDir);
			}
			else if (mat.type == MaterialType::GLOSSY)
			{
				// Monte Carlo Integration
				glm::vec3 u = fabs(n.x) < 1 - FLT_EPSILON ? glm::cross(glm::vec3(1, 0, 0), r) : glm::cross(glm::vec3(1), r);
				u = glm::normalize(u);
				glm::vec3 v = glm::cross(u, r);
				float w = hiprand_uniform(&state) * mat.roughness, theta = hiprand_uniform(&state);
				// wighted sampling on hemisphere
				reflectDir = w * cosf(2 * PI * theta) * u + w * sinf(2 * PI * theta) * v + sqrtf(1 - w * w) * r;
			}
			else if (mat.type == MaterialType::GLASS)
			{
				float nc = 1.0f, ng = 1.5f;
				// Snells law
				float eta = inside ? ng / nc : nc / ng;
				float r0 = (nc - ng) / (nc + ng);
				r0 = r0 * r0;
				float c = fabs(glm::dot(rd, n));
				float k = 1.0f - eta * eta * (1.0f - c * c);
				if (k < 0.0f)
					reflectDir = r;
				else
				{
					// Shilick's approximation of Fresnel's equation
					float re = r0 + (1.0f - r0) * (1.0f - c) * (1.0f - c);
					if (fabs(hiprand_uniform(&state)) < re)
						reflectDir = r;
					else
					{
						reflectDir = glm::normalize(eta * rd - (eta * glm::dot(n, rd) + sqrtf(k)) * n);
						p -= n * EPSILON * 2.0f;
						inside = !inside;
					}
				}
			}

			polarInfoList[depth - 1].indir = rd;
			polarInfoList[depth - 1].outdir = reflectDir;
			polarInfoList[depth - 1].normal = glm::normalize(reflectDir - rd);
			if (fabs(1.0f - glm::dot(reflectDir, rd)) < EPSILON)
				polarInfoList[depth - 1].normal = n;
			polarInfoList[depth - 1].R = GetRsRp(1.0f, mat.ior, glm::dot(polarInfoList[depth - 1].normal, rd));
			float theta = 2.0f * PI * hiprand_uniform(&state);
			float intensity = mat.intensity;
			if (mat.intensityTexId != -1)
			{
				// TODO map intensity value here
				intensity = CUDATex2D(textures[mat.intensityTexId], uv).x;
			}
			polarInfoList[depth - 1].S = glm::vec3(1.0f, cosf(theta), sinf(theta)) * intensity;
			
			return mat.emissive * mat.emissiveIntensity + Trace(p, reflectDir, depth, inside, state, polarInfoList) * mat.baseColor;
		}
	}

	return glm::vec3(0.0f);
}

__global__ void RenderPixel(float* img, float* data)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= resX || y >= resY)
		return;
	const int index = x + (resY - y - 1)*resX;
	hiprandState_t localState = state[index];

	// Position world space image plane
	glm::vec3 camPos = glm::vec3(_camPos[0], _camPos[1], _camPos[2]);
	glm::vec3 camDir = glm::vec3(_camDir[0], _camDir[1], _camDir[2]);
	glm::vec3 camUp = glm::vec3(_camUp[0], _camUp[1], _camUp[2]);
	glm::vec3 imgCenter = camPos + camDir * camFocal;
	float imgHeight = 2.0f * camFocal * tan((camFovy / 2.0f) * PI / 180.0f);
	float aspect = (float)resX / (float)resY;
	float imgWidth = imgHeight * aspect;
	float deltaX = imgWidth / (float)resX;
	float deltaY = imgHeight / (float)resY;
	glm::vec3 camRight = glm::normalize(glm::cross(camUp, camDir));

	// Starting at top left
	glm::vec3 topLeft = imgCenter - camRight * (imgWidth * 0.5f);
	topLeft += camUp * (imgHeight * 0.5f);
	glm::vec3 pixel = topLeft - camUp * (float(y) * deltaY) + camRight * (float(x) * deltaX);

	/* ----- GATHER POLAR INFO ----- */
	GPUPolarInfo polarInfoList[10];
	/* ----- GATHER POLAR INFO ----- */

	glm::vec3 rayDir = glm::normalize(pixel - camPos);
	int depth = 0;
	bool inside = false;
	glm::vec3 color = Trace(camPos, rayDir, depth, inside, localState, polarInfoList);

	/* ----- CALCULAT POLAR RESULT ----- */
	glm::vec3 polarResult = CalculatePolarResult(rayDir, polarInfoList, depth, localState);
	if (isnan(polarResult.x) || isnan(polarResult.y) || isnan(polarResult.z))
		polarResult = glm::vec3(0.0f);
	color = polarResult;
	/* ----- CALCULAT POLAR RESULT ----- */

	// Draw
	glm::vec3 colorR = color * 5.0f;
	glm::vec3 colorGB = 1.0f - (color + 0.025f) * 20.0f;
	color = glm::vec3(colorR.x, colorGB.y, colorGB.z);

	glm::vec3 preColor = glm::vec3();
	memcpy(&preColor[0], img + 3 * index, 3 * sizeof(float));
	color = (preColor * float(samples - 1) + color) / float(samples);
	color = glm::clamp(color, glm::vec3(0.0f), glm::vec3(1.0f));
	memcpy(img + 3 * index, &color[0], 3 * sizeof(float));

	// Data
	glm::vec3 preRes = glm::vec3();
	memcpy(&preRes[0], data + 3 * index, 3 * sizeof(float));
	polarResult = (preRes * float(samples - 1) + polarResult) / float(samples);
	memcpy(data + 3 * index, &polarResult[0], 3 * sizeof(float));

	state[index] = localState;
}

void CUDARenderFrame(int w, int h, float* img, float* data, int& h_samples)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(samples), &h_samples, sizeof(int)));
	dim3 blockDim(16, 16, 1), gridDim(w / blockDim.x + 1, h / blockDim.y + 1, 1);
	RenderPixel << <gridDim, blockDim >> > (img, data);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void CUDAReset()
{
	if (bvh != 0)
	{
		gpuErrchk(hipFree(bvh));
		bvh = 0;
	}

	if (textures != 0)
	{
		for (int i = 0; i < numTextures; i++)
			gpuErrchk(hipFree(textures[i].data));
		gpuErrchk(hipFree(textures));
		textures = 0;
		numTextures = 0;
	}
}

void CUDAFinish()
{
	gpuErrchk(hipFree(state));
}
